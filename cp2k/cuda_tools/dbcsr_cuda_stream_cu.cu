/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013 CP2K developers group
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "dbcsr_cuda.h"
#include <math.h>

  static const int verbose_print = 0;

extern "C" int cuda_stream_create(hipStream_t** stream_p){
  if(verbose_print) printf("cuda_stream_create called\n");
  *stream_p = (hipStream_t*) malloc(sizeof(hipStream_t));
  hipError_t cErr = hipStreamCreate(*stream_p);
  if (cuda_error_check(cErr)) return 1;
  if (cuda_error_check(hipGetLastError())) return 1;
  return 0;
}

extern "C" int cuda_stream_destroy(hipStream_t* stream){
    if(verbose_print) printf("cuda_stream_destroy called\n");
    hipError_t cErr = hipStreamDestroy(*stream);
    free(stream);
    if (cuda_error_check (cErr)) return 1;
    if (cuda_error_check(hipGetLastError ()))return 1;
    return 0;
}

extern "C" int cuda_stream_sync(hipStream_t* stream)
{
  hipError_t cErr;
  cErr = hipStreamSynchronize(*stream);
  if (cuda_error_check (cErr))
    return 1;
  return 0;
}
