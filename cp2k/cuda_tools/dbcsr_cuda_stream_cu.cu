/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013 CP2K developers group
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "dbcsr_cuda.h"
#include <math.h>

#ifdef __CUDA_PROFILING
#include <nvToolsExtCudaRt.h>
#endif

  static const int verbose_print = 0;

extern "C" int cuda_stream_create(hipStream_t** stream_p){
  *stream_p = (hipStream_t*) malloc(sizeof(hipStream_t));
  hipError_t cErr = hipStreamCreate(*stream_p);
  if(verbose_print) printf("cuda_stream_create: %p -> %d \n", *stream_p, **stream_p);
  if (cuda_error_check(cErr)) return 1;
  if (cuda_error_check(hipGetLastError())) return 1;
  return 0;
}

#ifndef __HAS_NO_CUDA_STREAM_PRIORITIES
extern "C" int cuda_stream_create_with_priority(hipStream_t** stream_p, int priority){
  if(verbose_print) printf("cuda_stream_create_with_priority called\n");
  *stream_p = (hipStream_t*) malloc(sizeof(hipStream_t));
  unsigned int flags = hipStreamNonBlocking;
  hipError_t cErr =  hipStreamCreateWithPriority(*stream_p, flags, priority);
  if (cuda_error_check(cErr)) return 1;
  if (cuda_error_check(hipGetLastError())) return 1;
  return 0;
}

extern "C" int cuda_stream_priority_range(int* least, int* greatest){
  hipError_t cErr = hipDeviceGetStreamPriorityRange(least, greatest);
  if (cuda_error_check(cErr)) return 1;
  if (cuda_error_check(hipGetLastError())) return 1;
  return 0;
}
#endif

extern "C" int cuda_stream_destroy(hipStream_t* stream){
    if(verbose_print) printf("cuda_stream_destroy called\n");
    hipError_t cErr = hipStreamDestroy(*stream);
    free(stream);
    if (cuda_error_check (cErr)) return 1;
    if (cuda_error_check(hipGetLastError ()))return 1;
    return 0;
}

extern "C" int cuda_stream_sync(hipStream_t* stream)
{
  hipError_t cErr;
  cErr = hipStreamSynchronize(*stream);
  if (cuda_error_check (cErr))
    return 1;
  return 0;
}

#ifdef __CUDA_PROFILING
extern "C" void cuda_stream_set_name(hipStream_t* stream_p, char* name){
  nvtxNameCudaStreamA(*stream_p, name);
}
#endif
