#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/types.h>
#include <unistd.h>

extern void cuda_error_check2 (hipError_t hipError_t, int line) {
  int         pid;
  size_t      free, total;
  hipError_t cErr2;

  cErr2 = hipGetLastError();
  if (hipError_t != hipSuccess || cErr2 != hipSuccess) {
    pid = getpid();
    printf("%d CUDA RT Error line %d\n", pid, line);
    printf("%d CUDA RT1 Error: %s\n", pid, hipGetErrorString(hipError_t));
    printf("%d CUDA RT2 Error: %s\n", pid, hipGetErrorString(cErr2));
    hipMemGetInfo(&free,&total);
    printf("%d Free: %zu , Total: %zu\n", pid, free, total);
    fflush(stdout);
    exit(-1);
  }
}


#if defined ( __PW_CUDA )
#include <cufft.h>

extern void cufft_error_check2 (cufftResult_t cufftError, int line) {
  int         pid;
  size_t      free, total;
  cudaError_t cErr2;

  cErr2 = cudaGetLastError();
  if (cufftError != CUFFT_SUCCESS || cErr2 != cudaSuccess) {
    pid = getpid();
    printf("%d CUDA FFT Error line: %d \n", pid, line);
    switch (cufftError) {
      case CUFFT_INVALID_PLAN:   printf("%d CUDA FFT1 Error (CUFFT_INVALID_PLAN)\n", pid); break;
      case CUFFT_ALLOC_FAILED:   printf("%d CUDA FFT1 Error (CUFFT_ALLOC_FAILED)\n", pid); break;
      case CUFFT_INVALID_VALUE:  printf("%d CUDA FFT1 Error (CUFFT_INVALID_VALUE)\n", pid); break;
      case CUFFT_INTERNAL_ERROR: printf("%d CUDA FFT1 Error (CUFFT_INTERNAL_ERROR)\n", pid); break;
      case CUFFT_EXEC_FAILED:    printf("%d CUDA FFT1 Error (CUFFT_EXEC_FAILED)\n", pid); break;
      case CUFFT_INVALID_SIZE:   printf("%d CUDA FFT1 Error (CUFFT_INVALID_SIZE)\n", pid); break;
      default: printf("%d CUDA FFT1 Error (--unimplemented--) %d %d\n", pid, cufftError, cErr2); break;
    }
    printf("%d CUDA FFT2 Error %s \n", pid, cudaGetErrorString(cErr2));
    cudaMemGetInfo(&free,&total);
    printf("%d Free: %zu , Total: %zu\n", pid, free, total);
    fflush(stdout);
    exit(-1);
  }
}

#endif
