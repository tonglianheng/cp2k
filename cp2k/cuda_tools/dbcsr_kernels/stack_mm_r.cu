#include "hip/hip_runtime.h"
/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013  Urban Borstnik and the CP2K developers group
 *****************************************************************************/

// DBCSR_KERNEL datatype=dbcsr_type_real_4, homogeneous_only=False

#include "dbcsr_kernel.h"
#include "stack_mm_r.h"

extern __shared__ double cache[];

//==============================================================================
__global__ void stack_mm_r
  (const int *__restrict__ param_stack,
   int stack_size,
   const float *__restrict__ a_data,
   const float *__restrict__ b_data,
   float *__restrict__ c_data)
{

  /**
   *  \var sp        which stack member this thread block is processing
                     (= CUDA thread block)
   *  \var sp_one    translated stack (=sp+1)
   *  \var tn        thread number (of CUDA thread block)
   *  \var nt        number of threads (size of CUDA thread block)
   *  \var our_params  cache for this thread block's multiplication parameters
   *  \var m, n, k   dimensions of the blocks (C is m*n, A is m*k, B is k*n)
   *  \var mn, mk, kn  product of the block dimensions
   *  \var l         multiplication loop index
   *  \var c, r      C matrix row, column of this thread
   *  \var myc       C matrix accumulator
   *  \var buff      cache for A and B data
   *  \var c_id      translated C block number (used in locking)
   *  \var lock_owner  current C block owner (used in locking)
   */

  int sp;			//, lock_owner, c_id, sp_one;
  int tn, nt;
  int r, c, l;
  int m, n, k;
  int mn, mk, kn;
  float myc;
  __shared__ int our_params[7];
  float *buff;


  /* Setup shared memory. */
  buff = (float *) cache;

  /* Determine who I am. */
  sp = blockIdx.x;
  tn = threadIdx.x;
  nt = blockDim.x;

  /* Load in the parameters. */
  for (l = 0; l <= 6 / nt; l++)
    {
      r = tn + nt * l;
      if (r < 7)
	our_params[r] = param_stack[7 * sp + r];
    }

  syncthreads ();
  m = our_params[0];
  n = our_params[1];
  k = our_params[2];

  /* Load in the buffers. */
  mk = m * k;
  kn = k * n;
  for (l = 0; l <= (mk - 1) / nt; l++)
    {
      r = tn + nt * l;
      if (r < mk)
	buff[r] = a_data[our_params[3] - 1 + r];
    }
  for (l = 0; l <= (kn - 1) / nt; l++)
    {
      r = tn + nt * l;
      if (r < kn)
	buff[mk + r] = b_data[our_params[4] - 1 + r];
    }

  /* Calculate who I am. */
  syncthreads ();

  mn = m * n;

  /* Do actual multiplication. */
  if (tn < mn)
    {
      r = tn % m;
      c = tn / m;
      myc = 0.0f;

      for (l = 0; l < k; l++)
	{
	  myc = myc + buff[l * m + r] * buff[mk + c * k + l];
	}
    }

  /* Add results to global C block. */
  if (tn < mn)
    atomicAdd (&c_data[our_params[5] - 1 + tn], myc);

}


//==============================================================================
int launch_stack_mm_r(int *param_stack, int stack_size, hipStream_t stream,
    int m_max, int n_max, int k_max,
    float *a_data, float *b_data, float *c_data){

     int shared_size = (m_max * k_max + k_max * n_max) * sizeof (float);
     if (shared_size > devProperties.sharedMemPerBlock)
         return 4;

     int maxt = m_max * n_max;
     if (maxt > devProperties.maxThreadsPerBlock)
         return 3;

     stack_mm_r <<< stack_size, maxt, shared_size, stream >>>
        (param_stack, stack_size, a_data, b_data, c_data);
     return(0);
}
