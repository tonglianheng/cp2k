#include "hip/hip_runtime.h"
/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013  Urban Borstnik and the CP2K developers group
 *****************************************************************************/

// DBCSR_KERNEL datatype=dbcsr_type_complex_8, homogeneous_only=False

#include "dbcsr_kernel.h"
#include "stack_mm_z.h"

extern __shared__ double cache[];

//==============================================================================
__global__ void stack_mm_z
  (const int *__restrict__ param_stack,
   int stack_size,
   const double *__restrict__ a_data,
   const double *__restrict__ b_data,
   double *__restrict__ c_data)
{

  /**
   *  \var sp        which stack member this thread block is processing
                     (= CUDA thread block)
   *  \var sp_one    translated stack (=sp+1)
   *  \var tn        thread number (of CUDA thread block)
   *  \var nt        number of threads (size of CUDA thread block)
   *  \var our_params  cache for this thread block's multiplication parameters
   *  \var m, n, k   dimensions of the blocks (C is m*n, A is m*k, B is k*n)
   *  \var mn, mk, kn  product of the block dimensions
   *  \var l         multiplication loop index
   *  \var c, r      C matrix row, column of this thread
   *  \var myc       C matrix accumulator
   *  \var buff      cache for A and B data
   *  \var c_id      translated C block number (used in locking)
   *  \var lock_owner  current C block owner (used in locking)
   */

  int sp;			//, lock_owner, c_id, sp_one;
  int tn, nt;
  int r, c, l;
  int m, n, k;
  int mn, mk, kn;
  double myc_r, myc_i;
  __shared__ int our_params[7];
  double *buff;


  /* Setup shared memory. */
  buff = (double *) cache;

  /* Determine who I am. */
  sp = blockIdx.x;
  tn = threadIdx.x;
  nt = blockDim.x;

  /* Load in the parameters. */
  for (l = 0; l <= 6 / nt; l++)
    {
      r = tn + nt * l;
      if (r < 7)
	our_params[r] = param_stack[7 * sp + r];
    }

  syncthreads ();
  m = our_params[0];
  n = our_params[1];
  k = our_params[2];

  /* Load in the buffers. */
  mk = 2 * m * k;
  kn = 2 * k * n;
  for (l = 0; l <= (mk - 1) / nt; l++)
    {
      r = tn + nt * l;
      if (r < mk)
	{
	  buff[r] = a_data[2 * (our_params[3] - 1) + r];
	}
    }
  for (l = 0; l <= (kn - 1) / nt; l++)
    {
      r = tn + nt * l;
      if (r < kn)
	{
	  buff[mk + r] = b_data[2 * (our_params[4] - 1) + r];
	}
    }
  mk = m * k;
  kn = k * n;

  /* Calculate who I am. */
  syncthreads ();

  mn = m * n;

  /* Do actual multiplication. */
  if (tn < mn)
    {
      r = tn % m;
      c = tn / m;
      myc_r = 0.0l;
      myc_i = 0.0l;

      for (l = 0; l < k; l++)
	{
	  myc_r = myc_r +
	    buff[2 * (l * m + r)] *
	    buff[2 * (mk + c * k + l)] -
	    buff[2 * (l * m + r) + 1] * buff[2 * (mk + c * k + l) + 1];
	  myc_i = myc_i +
	    buff[2 * (l * m + r)] *
	    buff[2 * (mk + c * k + l) + 1] +
	    buff[2 * (l * m + r) + 1] * buff[2 * (mk + c * k + l)];
	}
    }

  /* Add results to global C block. */
  if (tn < mn)
    {
      atomicAdd (&c_data[2 * (our_params[5] - 1 + tn)], myc_r);
      atomicAdd (&c_data[2 * (our_params[5] - 1 + tn) + 1], myc_i);
    }

}


//==============================================================================
int launch_stack_mm_z(int *param_stack, int stack_size, hipStream_t stream,
    int m_max, int n_max, int k_max,
    double *a_data, double *b_data, double *c_data){

     int shared_size = (m_max * k_max + k_max * n_max) * sizeof (double) * 2;
     if (shared_size > devProperties.sharedMemPerBlock)
          return 4;

     int maxt = m_max * n_max;
     if (maxt > devProperties.maxThreadsPerBlock)
         return 3;

     stack_mm_z <<< stack_size, maxt, shared_size, stream >>>
        (param_stack, stack_size, a_data, b_data, c_data);
     return(0);
}

