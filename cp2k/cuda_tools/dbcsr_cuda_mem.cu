/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013  Urban Borstnik and the CP2K developers group
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "dbcsr_cuda.h"
#include <math.h>


static const int verbose_print = 0;


//==============================================================================
extern "C" int
dc_dev_mem_alloc (void **dev_mem, size_t n)
{
  hipError_t cErr;

  cErr = hipMalloc ((void **) dev_mem, (size_t) n);
  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;
  if (dev_mem == NULL)
    return 2;
  if (verbose_print)
    printf ("Device allocation address %p, size %ld\n", *dev_mem, (long) n);

  return 0;
}

/*
//==============================================================================
extern "C" int
dc_dev_mem_realloc (void **dev_mem, size_t n, size_t old_n,
		    int *memory_crunch)
{
  hipError_t cErr;
  void *new_dev_mem;
  size_t count;

  *memory_crunch = 0;
  cErr = hipMalloc ((void **) &new_dev_mem, (size_t) n);
  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;
  if (new_dev_mem == NULL)
    return 2;
  if (verbose_print)
    printf ("Device allocation address %p, size %ld\n", new_dev_mem,
	    (long) n);
  count = MIN (old_n, n);
  if (count > 0)
    {
      if (verbose_print)
	printf ("Copy %d bytes.\n", (int) count);
      cErr =
	hipMemcpy (new_dev_mem, *dev_mem, count, hipMemcpyDeviceToDevice);
      if (cuda_error_check (cErr))
	return 1;
      if (cuda_error_check (hipGetLastError ()))
	return 1;
    }

  cErr = hipFree ((void *) *dev_mem);
  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;

  *dev_mem = new_dev_mem;
  return 0;
}
*/

//==============================================================================
extern "C" int
dc_dev_mem_dealloc (void *dev_mem)
{
  hipError_t cErr;

  if (verbose_print)
    printf ("Device deallocation address %p\n", dev_mem);
  cErr = hipFree ((void *) dev_mem);
  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;

  return 0;
}

//==============================================================================
extern "C" int
dc_host_mem_alloc (void **host_mem, size_t n, int wc, int port)
{
  hipError_t cErr;
  unsigned int flag;

  flag = hipHostMallocDefault;
  if (wc)
    flag |= hipHostMallocWriteCombined;
  if (port)
    flag |= hipHostMallocPortable;
  cErr = hipHostAlloc ((void **) host_mem, (size_t) n, flag);
  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;
  if (host_mem == NULL)
    return 2;
  if (verbose_print)
    printf ("Allocating %d bytes of host pinned memory at %p\n",n,  *host_mem);

  return 0;
}

//==============================================================================
extern "C" int
dc_host_mem_dealloc (void *host_mem)
{
  hipError_t cErr;

  if (verbose_print)
    printf ("Host pinned deallocation address %p\n", host_mem);
  cErr = hipHostFree ((void *) host_mem);
  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;

  return 0;
}

//==============================================================================
extern "C" int
dc_memcpy_h2d_cu (const void *host_mem, void *dev_mem, size_t count, hipStream_t* stream)
{
  hipError_t cErr;

  if (verbose_print)
      printf ("Copyint %d bytes from host address %p to device address %p \n",count, host_mem, dev_mem);

  cErr = hipMemcpyAsync (dev_mem, host_mem, count, hipMemcpyHostToDevice, *stream);

  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;

  return 0;
}

//==============================================================================
extern "C" int
dc_memcpy_d2h_cu (const void *dev_mem, void *host_mem, size_t count, hipStream_t* stream)
{
  hipError_t cErr;

  if (verbose_print)
      printf ("Copying %d bytes from device address %p to host address %p\n", count, dev_mem, host_mem);

  cErr = hipMemcpyAsync (host_mem, dev_mem, count, hipMemcpyDeviceToHost, *stream);

  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;
  if (verbose_print)
    printf ("d2h %f\n", *((double *) host_mem));

  return 0;
}

//==============================================================================
extern "C" int
dc_memcpy_d2d_cu (const void *devmem_src, void *devmem_dst, size_t count, hipStream_t* stream)
{
  hipError_t cErr;

  if (verbose_print)
      printf ("Coping %d bytes from device address %p to device address %p \n", count, devmem_src, devmem_dst);


  if(stream == NULL){
      cErr = hipMemcpy (devmem_dst, devmem_src, count, hipMemcpyDeviceToDevice);
  }else{
      cErr = hipMemcpyAsync (devmem_dst, devmem_src, count, hipMemcpyDeviceToDevice, *stream);
  }

  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;

  return 0;
}

//==============================================================================
extern "C" int
dc_memzero_cu (void *dev_mem, size_t offset, size_t length, hipStream_t* stream)
{
  hipError_t cErr;
  if(stream == NULL){
      cErr = hipMemset ((void *) (((char *) dev_mem) + offset), (int) 0, length);
  }else{
      cErr = hipMemsetAsync ((void *) (((char *) dev_mem) + offset), (int) 0, length, *stream);
  }

  if (verbose_print)
    printf ("Zero at device address %p, offset %d, len %d\n",
     dev_mem, (int) offset, (int) length);
  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;

  return 0;
}

//==============================================================================
extern "C" int
dc_dev_mem_info_cu (size_t * free, size_t * avail)
{
  hipError_t cErr;
  cErr = hipMemGetInfo (free, avail);
  if (cuda_error_check (cErr))
    return 1;
  if (cuda_error_check (hipGetLastError ()))
    return 1;
  return 0;
}
