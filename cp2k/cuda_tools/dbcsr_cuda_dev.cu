/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013  Urban Borstnik and the CP2K developers group
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sm_11_atomic_functions.h>

#include "dbcsr_cuda.h"


struct hipDeviceProp_t devProperties;
#pragma omp threadprivate(devProperties)


int cuda_error_check (hipError_t hipError_t){
  if (hipError_t != hipSuccess){
      printf ("CUDA Error: %s\n", hipGetErrorString (hipError_t));
      return 1;
    }
  return 0;
};


extern "C" int
dc_device_sync_cu ()
{
  hipError_t cErr;

  cErr = hipDeviceSynchronize ();
  if (cuda_error_check (cErr))
    return 1;
  return 0;
}


extern "C" int
dc_set_device_cu (int device_id)
{
  hipError_t cErr;
  int myDevice;

  cErr = hipSetDevice (device_id);
  if (cuda_error_check (cErr))
    return 1;

  cErr = hipGetDevice (&myDevice);
  if (cuda_error_check (cErr))
    return 1;

  if (myDevice != device_id)
    return 1;

  cErr = hipGetDeviceProperties (&devProperties, myDevice);
  if (cuda_error_check (cErr))
    return 1;

  return 0;
}

extern "C" int
dc_get_ndevices_cu (int *n_devices)
{
  hipError_t cErr;

  cErr = hipGetDeviceCount (n_devices);
  if (cuda_error_check (cErr))
    return 1;
  return 0;
}

extern "C" int dc_set_shared_mem_config(int config){
  if(config==0)
      return hipDeviceSetSharedMemConfig(hipSharedMemBankSizeDefault);
  if(config==1)
      return hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
  if(config==2)
      return hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
  return(-1);
}

