#include "hip/hip_runtime.h"
/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013  Urban Borstnik and the CP2K developers group
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "dbcsr_cuda.h"
#include <math.h>


  static const int verbose_print = 0;


extern "C" int cuda_event_create(hipEvent_t** event_p){
  *event_p = (hipEvent_t*) malloc(sizeof(hipEvent_t));
  hipError_t cErr = hipEventCreate(*event_p);
  if(verbose_print) printf("cuda_event_created:  %p -> %d\n", *event_p, **event_p);
  if (cuda_error_check(cErr)) return 1;
  if (cuda_error_check(hipGetLastError())) return 1;
  return 0;
}

extern "C" int cuda_event_destroy(hipEvent_t* event){
    if(verbose_print) printf("cuda_event_destroy called\n");
    hipError_t cErr = hipEventDestroy(*event);
    free(event);
    if (cuda_error_check (cErr)) return 1;
    if (cuda_error_check(hipGetLastError ()))return 1;
    return 0;
}

extern "C" int cuda_event_record(hipEvent_t* event, hipStream_t* stream){
    if(verbose_print) printf("cuda_event_record: %p -> %d,  %p -> %d\n", event, * event,  stream, *stream);
    hipError_t cErr = hipEventRecord (*event, *stream);
    if (cuda_error_check (cErr)) return 1;
    //if (cuda_error_check(hipGetLastError ()))return 1;
    return 0;
}

extern "C" int cuda_event_query(hipEvent_t* event){
    if(verbose_print) printf("cuda_event_query called\n");
    hipError_t cErr = hipEventQuery(*event);
    //if(cuda_error_check(hipGetLastError ())) return -1;
    if(cErr==hipSuccess) return 0;
    if(cErr==hipErrorNotReady) return 1;
    return -2;
}

extern "C" int cuda_stream_wait_event(hipStream_t* stream, hipEvent_t* event){
    if(verbose_print) printf("cuda_stream_wait_event called\n");
    // flags: Parameters for the operation (must be 0)
    hipError_t cErr = hipStreamWaitEvent(*stream, *event, 0);
    if (cuda_error_check (cErr)) return 1;
    //if (cuda_error_check(hipGetLastError ()))return 1;
    return 0;
}

extern "C" int cuda_event_synchronize(hipEvent_t* event){
    if(verbose_print) printf("cuda_event_synchronize called\n");
    hipError_t cErr = hipEventSynchronize(*event);
    if (cuda_error_check (cErr)) return 1;
    if (cuda_error_check(hipGetLastError ()))return 1;
    return 0;
}

//
// extern "C" int
// dc_dev_mem_realloc (void **dev_mem, size_t n, size_t old_n,
// 		    int *memory_crunch)
// {
//   hipError_t cErr;
//   void *new_dev_mem;
//   size_t count;
// 
//   *memory_crunch = 0;
//   cErr = hipMalloc ((void **) &new_dev_mem, (size_t) n);
//   if (cuda_error_check (cErr))
//     return 1;
//   if (cuda_error_check (hipGetLastError ()))
//     return 1;
//   if (new_dev_mem == NULL)
//     return 2;
//   if (verbose_print)
//     printf ("Device allocation address %p, size %ld\n", new_dev_mem,
// 	    (long) n);
//   count = MIN (old_n, n);
//   if (count > 0)
//     {
//       if (verbose_print)
// 	printf ("Copy %d bytes.\n", (int) count);
//       cErr =
// 	hipMemcpy (new_dev_mem, *dev_mem, count, hipMemcpyDeviceToDevice);
//       if (cuda_error_check (cErr))
// 	return 1;
//       if (cuda_error_check (hipGetLastError ()))
// 	return 1;
//     }
// 
//   cErr = hipFree ((void *) *dev_mem);
//   if (cuda_error_check (cErr))
//     return 1;
//   if (cuda_error_check (hipGetLastError ()))
//     return 1;
// 
//   *dev_mem = new_dev_mem;
//   return 0;
// }
// 
// extern "C" int
// dc_dev_mem_dealloc (void *dev_mem)
// {
//   hipError_t cErr;
// 
//   if (verbose_print)
//     printf ("Device deallocation address %p\n", dev_mem);
//   cErr = hipFree ((void *) dev_mem);
//   if (cuda_error_check (cErr))
//     return 1;
//   if (cuda_error_check (hipGetLastError ()))
//     return 1;
// 
//   return 0;
// }
// 
// extern "C" int
// dc_host_mem_alloc (void **host_mem, size_t n, int wc, int port)
// {
//   hipError_t cErr;
//   unsigned int flag;
// 
//   flag = hipHostMallocDefault;
//   if (wc)
//     flag |= hipHostMallocWriteCombined;
//   if (port)
//     flag |= hipHostMallocPortable;
//   cErr = hipHostAlloc ((void **) host_mem, (size_t) n, flag);
//   if (cuda_error_check (cErr))
//     return 1;
//   if (cuda_error_check (hipGetLastError ()))
//     return 1;
//   if (host_mem == NULL)
//     return 2;
//   if (verbose_print)
//     printf ("Host pinned allocation address %p\n", *host_mem);
// 
//   return 0;
// }
// 
// extern "C" int
// dc_host_mem_dealloc (void *host_mem)
// {
//   hipError_t cErr;
// 
//   if (verbose_print)
//     printf ("Host pinned deallocation address %p\n", host_mem);
//   cErr = hipHostFree ((void *) host_mem);
//   if (cuda_error_check (cErr))
//     return 1;
//   if (cuda_error_check (hipGetLastError ()))
//     return 1;
// 
//   return 0;
// }
// 
// 
// extern "C" int
// dc_memcpy_h2d_cu (const void *host_mem, void *dev_mem, size_t count,
// 		  int async_type, int stream_id)
// {
//   hipError_t cErr;
// 
//   if (verbose_print)
//     {
//       printf ("Copy from host address %p\n", host_mem);
//       printf ("Copy to device address %p\n", dev_mem);
//       printf ("h2d %f\n", *((double *) host_mem));
//       printf ("Async? %d\n", async_type);
//     }
// 
//   switch (async_type)
//     {
//     case 0:
//       /* Synchronous */
//       cErr = hipMemcpy (dev_mem, host_mem, count, hipMemcpyHostToDevice);
//       break;
//     case 1:
//       /* Asynchronous */
//       cErr =
// 	hipMemcpyAsync (dev_mem, host_mem, count, hipMemcpyHostToDevice,
// 			 (hipStream_t) dc_get_stream (stream_id));
//       break;
//     case 2:
//       cErr =
// 	hipMemcpyAsync (dev_mem, host_mem, count, hipMemcpyHostToDevice,
// 			 (hipStream_t) dc_get_stream (stream_id));
//       /* Try async if sync is unsuccessful. */
//       if (cuda_error_check (cErr))
// 	{
// 	  if (verbose_print)
// 	    printf ("Async unsuccessful, trying sync.\n");
// 	  cErr =
// 	    hipMemcpy (dev_mem, host_mem, count, hipMemcpyHostToDevice);
// 	}
//       break;
//     }
//   if (cuda_error_check (cErr))
//     return 1;
//   if (cuda_error_check (hipGetLastError ()))
//     return 1;
// 
//   return 0;
// }
// 
// 
// extern "C" int
// dc_memcpy_d2h_cu (const void *dev_mem, void *host_mem, size_t count,
// 		  int async_type, int stream_id)
// {
//   hipError_t cErr;
// 
//   if (verbose_print)
//     {
//       printf ("Copy from device address %p\n", dev_mem);
//       printf ("Copy to host address %p\n", host_mem);
//       printf ("Async? %d\n", async_type);
//     }
//   switch (async_type)
//     {
//     case 0:
//       /* Synchronous */
//       cErr = hipMemcpy (host_mem, dev_mem, count, hipMemcpyDeviceToHost);
//       break;
//     case 1:
//       /* Asynchronous */
//       cErr =
// 	hipMemcpyAsync (host_mem, dev_mem, count, hipMemcpyDeviceToHost,
// 			 (hipStream_t) dc_get_stream (stream_id));
//       break;
//     case 2:
//       cErr =
// 	hipMemcpyAsync (host_mem, dev_mem, count, hipMemcpyDeviceToHost,
// 			 (hipStream_t) dc_get_stream (stream_id));
//       /* Try async if sync is unsuccessful. */
//       if (cuda_error_check (cErr))
// 	{
// 	  if (verbose_print)
// 	    printf ("Async unsuccessful, trying sync.\n");
// 	  cErr =
// 	    hipMemcpy (host_mem, dev_mem, count, hipMemcpyDeviceToHost);
// 	}
//       break;
//     }
//   if (cuda_error_check (cErr))
//     return 1;
//   if (cuda_error_check (hipGetLastError ()))
//     return 1;
//   if (verbose_print)
//     printf ("d2h %f\n", *((double *) host_mem));
// 
//   return 0;
// }
// 
// 
// extern "C" int
// dc_memzero_cu (void *dev_mem, size_t offset, size_t length)
// {
//   hipError_t cErr;
// 
//   cErr = hipMemset ((void *) (((char *) dev_mem) + offset), (int) 0, length);
//   if (verbose_print)
//     printf ("Zero at device address %p, offset %d, len %d\n",
// 	    dev_mem, (int) offset, (int) length);
//   if (cuda_error_check (cErr))
//     return 1;
//   if (cuda_error_check (hipGetLastError ()))
//     return 1;
// 
//   /*  struct hipDeviceProp_t devProperties;
//      int myDevice, nt, nb, ws, maxt;
// 
//      cErr = hipGetDevice(&myDevice);
//      if (cuda_error_check (cErr)) return 1;
// 
//      cErr = hipGetDeviceProperties(&devProperties, myDevice);
//      if (cuda_error_check (cErr)) return 1;
// 
//      ws = devProperties.warpSize;
//      maxt = devProperties.maxThreadsPerBlock;
//      printf("count %d, ws %d, maxt %d", (int) count, ws, maxt);
// 
//      nt = (int) sqrt(count);
//      nt = ((int) (nt + ws-1)/ws) * ws;
//      nt = MAX(MIN(nt, maxt), ws);
// 
//      printf("nt", nt);
// 
//      nb = (count+nt-1) / nt;
//      printf("nb", nb);
// 
// 
//      zeroMem <<< nb, nt >>> ((char *) dev_mem, (int) count);
//      if (cuda_error_check (hipGetLastError())) return 1; */
//   return 0;
// }
// 
// extern "C" int
// dc_dev_mem_info_cu (size_t * free, size_t * avail)
// {
//   hipError_t cErr;
//   cErr = hipMemGetInfo (free, avail);
//   if (cuda_error_check (cErr))
//     return 1;
//   if (cuda_error_check (hipGetLastError ()))
//     return 1;
//   return 0;
// }
