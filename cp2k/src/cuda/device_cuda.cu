/******************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations
 *  Copyright (C) 2000 - 2013 the CP2K developers group
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include "error_cuda.h"

//==============================================================================
extern "C" int cp_set_device_cu (int device_id)
{
  hipError_t cErr;
  int myDevice;

  cErr = hipSetDevice (device_id);
  cuda_error_check2(cErr, __LINE__);

  cErr = hipGetDevice (&myDevice);
  cuda_error_check2(cErr, __LINE__);

  if (myDevice != device_id)
    return 1;

  //cErr = hipGetDeviceProperties (&devProperties, myDevice);
  //cuda_error_check2(cErr, __LINE__);

  return 0;
}


//==============================================================================
extern "C" int cp_get_ndevices_cu (int *n_devices)
{
  hipError_t cErr;

  cErr = hipGetDeviceCount (n_devices);
  cuda_error_check2(cErr, __LINE__);
  return 0;
}


//==============================================================================
extern "C" int cp_device_sync_cu ()
{
  hipError_t cErr;

  cErr = hipDeviceSynchronize ();
  cuda_error_check2(cErr, __LINE__);
  return 0;
}


//==============================================================================
extern "C" int cp_device_reset_cu ()
{
  hipError_t cErr;

  cErr = hipDeviceReset();
  cuda_error_check2(cErr, __LINE__);
  return 0; 
}

//EOF
