/*****************************************************************************
 *  CP2K: A general program to perform molecular dynamics simulations        *
 *  Copyright (C) 2000 - 2014 the CP2K developers group                      *
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include "acc_cuda_error.h"
#include "../include/acc.h"

/****************************************************************************/
extern "C" int acc_get_ndevices(int *n_devices){
  hipError_t cErr;

  cErr = hipGetDeviceCount (n_devices);
  if (cuda_error_check (cErr))
    return -1;
  return 0;
}


/****************************************************************************/
extern "C" int acc_set_active_device(int device_id){
  hipError_t cErr;
  int myDevice;

  cErr = hipSetDevice (device_id);
  if (cuda_error_check (cErr))
    return -1;

  cErr = hipGetDevice (&myDevice);
  if (cuda_error_check (cErr))
    return -1;

  if (myDevice != device_id)
    return -1;

  return 0;
}

//EOF
